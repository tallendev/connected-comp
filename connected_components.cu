#include "hip/hip_runtime.h"
/*
 * Author: Tyler Allen
 * USAGE: ./concomp path-to-file
 *
 * Connected component finder. Works only on undirected matrices. Expects Matrix-Market format input file. Also expects
 * edges to have weights, due to sparse matrix library.
 *
 * Sparse matrix library is BeBOP SMC, included but not part of this project. I do not claim any of their code or 
 * library as my own.
 *
 */

#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <vector>
#include <iterator>
#include<unordered_set>
#include <assert.h>
#include "safecuda.h"

extern "C" {
#include "bebop/smc/csr_matrix.h"
#include "bebop/smc/sparse_matrix.h"
#include "bebop/smc/sparse_matrix_ops.h"
}

#define MAX_ITER (m)

/* 
 * GPU memory initialization kernel for connected components
 * Args:
 * m - from m x m matrix
 * I - buffer of length m+1 containing CSR-format rows
 * values - buffer of length M + 1 to be overwritten
 *
 * Values contains a unique global ID, which is just the row entry in the matrix.
 */
__global__ void init_cc(const int m, const int* const __restrict__ I, int* __restrict__ values)
{
    const int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < m + 1)
    {
        values[idx] = idx;
    }
}

/*
 *  connected components solving kernel
 *  Args:
 *  m, I, values are same as init_cc
 *  J - The column index array from CSR-format
 *
 *  each node corresponds to an index in the values array. At the beginning, all IDs are unique.
 *  Each iteration of this kernel, 1 thread is assigned to each node. This node replaces its value
 *  with the value of its lowest-value neighbor, including itself. After m calls to this kernel, all connected
 *  nodes within a single component will have the same minimum value. 
 */
__global__ void cudacc(const int m, const int* const __restrict__ I, const int* const __restrict__ J, int* __restrict__ values)
{
    int first_edge = 0;
    int num_edges = 0;
    const int idx = blockDim.x * blockIdx.x + threadIdx.x + 1;
    // in range
    if (idx < m + 1)
    {
        int minimum = values[idx];
        // not 0 edges
        if (I[idx] - I[idx - 1] > 0)
        {
            first_edge = I[idx - 1];
            num_edges = I[idx] - first_edge;
            minimum = values[idx];
            for (int j = first_edge; j < first_edge + num_edges; j++)
            {
                // find neighbor edge, see their true value
                minimum = min(minimum, values[J[j] + 1]);
            }
            // weak race condition. For huge matrices, may require some additional iterations
            // However, (m) iterations is already accounting worst-case behavior for the pathological list
            // network. In practice, this is far more than enough iterations. Also, thanks to
            // warp-synchronicity, the pathological list case likely will not fall into this issue either.
            // Therefore, this is a non-issue.
            values[idx] = minimum;
        }
    }
}

int main(int argc, char* argv[])
{
    if (argc != 2)
    {
        printf("Recv'd argc: %d\n", argc);
        exit(1);
    }
    char* filename = argv[1];
    // load mm
    struct sparse_matrix_t* mat = load_sparse_matrix(sparse_matrix_file_format_t::MATRIX_MARKET, filename);

    if (mat == nullptr)
    {
        printf("mat pointer: %p\nmaybe bad file name?\n", mat);
        exit(1);
    }
    // convert mm to csr
    int err = sparse_matrix_convert (mat, sparse_matrix_storage_format_t::CSR);
    
    // extract csr
    struct csr_matrix_t* cmat = (struct csr_matrix_t*) mat->repr;
    err = valid_csr_matrix_p (cmat);
    if (err == 0)
    {
        printf("not valid matrix before expansion\n");
        exit(1);
    }
    // expand to symmetry, assumed symmetric
    err = csr_matrix_expand_symmetric_storage (cmat);
    if (err != 0)
    {
        printf("some error expanding symmetric storage: %d\n", err);
        exit(1);
    }
    // makes sure matrix is sane
    err = valid_csr_matrix_p (cmat);
    if (err == 0)
    {
        printf("not valid matrix after expansion\n");
        exit(1);
    }
    int nnz = cmat->nnz;
    int m = cmat->m;
    int* I;
    int* values;
    printf("m, nnz: %d, %d\n", m, nnz);
    // map of real row indexes to sparse row indexes
    int* J;

    // alloc GPU memory for I, J, values defined in cuda section
    hipMalloc(&I, sizeof(int) * (m + 1));
    CHECK_CUDA_ERROR();
    hipMalloc(&J, sizeof(int) * nnz);
    CHECK_CUDA_ERROR();
    hipMalloc(&values, sizeof(int) * (m + 1));
    CHECK_CUDA_ERROR();

    // lets you know if you alloc too much memory : )
    printf("Sparse Structure requires %lu MB GPU memory\n", (sizeof(int) * nnz + 2 * sizeof(int) * (m + 1)) / 1000000);
    
    // copy sparse matrix into gpu memory
    hipMemcpy(I, cmat->rowptr, sizeof(int) * (m + 1), hipMemcpyHostToDevice);
    CHECK_CUDA_ERROR();
    hipMemcpy(J, cmat->colidx, sizeof(int) * nnz, hipMemcpyHostToDevice);
    CHECK_CUDA_ERROR();

    // init connected components algorithm
    init_cc<<<(m + 1) / 128 + 1, 128>>>(m, I, values);
    // repeat defined number of iterations
    for (int i = 0; i < MAX_ITER; i++)
    {
        cudacc<<<(m+1) / 128 + 1, 128>>>(cmat->m, I, J, values);
    }
    // wait for results
    hipDeviceSynchronize();
    CHECK_CUDA_ERROR();
    
    // copy result back
    int h_values[m + 1];
    hipMemcpy(h_values, values, sizeof(int) * (m + 1), hipMemcpyDeviceToHost);

    // count number of unique components via one list iteration
    std::unordered_set<int> uniques;
    for (int i = 1; i < m + 1; i++)
    {
        uniques.insert(h_values[i]);
    }
    printf("# of connected components: %lu\n", uniques.size());
    
    // clean house
    hipFree(I);
    hipFree(J);
    hipFree(values);
    destroy_sparse_matrix(mat);
    return 0;
}

